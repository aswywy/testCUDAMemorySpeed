#include "hip/hip_runtime.h"
#include "functions.h"
#include "functions.h"

//this one is wrong!!!!
__global__ void copyImagePitch(uchar * src, uchar * dst, uint imageH, uint imageW, uint channel)
{
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y + threadIdx.y;

	if ((x < imageW) && (y < imageH))
	{	
		uchar* ptr_src = src + 3* (y * imageW + x);
		uchar* ptr_dst = dst + 3* (y * imageW + x);
		/*
		ptr_dst[0] = (y * imageW + 3 * x) + 0;
		ptr_dst[1] = (y * imageW + 3 * x) + 1;
		ptr_dst[2] = (y * imageW + 3 * x) + 2;
		*/
		ptr_dst[0] = ptr_src[0];
		ptr_dst[1] = ptr_src[1];
		ptr_dst[2] = ptr_src[2];

	}
}
__global__ void copyImagePitch(uchar * src, uchar * dst, uint imageH, uint imageW, uint pitch ,uint channel)
{
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y + threadIdx.y;

	if ((x < imageW ) && (y < imageH))
	{
		if (channel == 3)
		{
			uchar* ptr_src = src + (y * pitch + 3 * x);
			uchar* ptr_dst = dst + (y * pitch + 3 * x);
			/*
			ptr_dst[0] = (y * pitch + 3 * x) + 0;
			ptr_dst[1] = (y * pitch + 3 * x) + 1;
			ptr_dst[2] = (y * pitch + 3 * x) + 2;
			*/
			ptr_dst[0] = ptr_src[0];
			ptr_dst[1] = ptr_src[1];
			ptr_dst[2] = ptr_src[2];
		}
		else ///channel == 4
		{
			uchar4* ptr_src = (uchar4*)(src + y * pitch) + x;
			uchar4* ptr_dst = (uchar4*)(dst + y * pitch) + x;
			ptr_dst[0] = ptr_src[0];
		}
		
	}
}

bool testGlobalReadPitch(cv::Mat &image, std::string  titile)
{
	uchar *devPtr_src, *devPtr_dst;
	size_t size_width = image.cols*sizeof(uchar)* image.channels();
	size_t size_height = image.rows;
	size_t size_pitch;
	
	cv::Mat dst_image;
	if (image.type() == CV_8UC3)
	{
		TIMED("malloc host dst memory")
		{
			dst_image.create(image.rows, image.cols, CV_8UC3);
		}
	}
	else if (image.type() == CV_8UC4)
	{
		TIMED("malloc host dst memory")
		{
			dst_image.create(image.rows, image.cols, CV_8UC4);
		}
	}
	TIMED("malloc device dst memory,and clear it!!")
	{
		checkCudaErrors(hipMallocPitch((void**)&devPtr_src,&size_pitch ,size_width,size_height));
		checkCudaErrors(hipMallocPitch((void**)&devPtr_dst, &size_pitch, size_width, size_height));
		checkCudaErrors(hipMemset2D(devPtr_dst, size_pitch, 0, size_width, size_height));
	}

	

	

	TIMED("hipMemcpyHostToDevice")
	{
		//hipPitchedPtr tep = make_hipPitchedPtr(image.data, size_pitch, size_width, size_height);
		checkCudaErrors(hipMemcpy2D(devPtr_src,size_pitch, image.data, size_width,size_width,size_height, hipMemcpyHostToDevice));
	}
	
	int width = image.cols;
	int height = image.rows;
	int channel = image.channels();

	dim3 blockSize(16, 16, 1);
	dim3 gridSize(((uint)width + blockSize.x - 1) / blockSize.x, ((uint)height + blockSize.y - 1) / blockSize.y, 1);
	//dim3 gridSize(((uint)size_pitch/3 + 1 + blockSize.x - 1) / blockSize.x, ((uint)height + blockSize.y - 1) / blockSize.y, 1);

	TIMED("copyImage")
	{
		for (int i = 0; i < 10000; i++)
		{
			//copyImagePitch << <gridSize, blockSize >> >(devPtr_src, devPtr_dst, (uint)height, (uint)width, (uint)channel);
			copyImagePitch << <gridSize, blockSize >> >(devPtr_src, devPtr_dst, (uint)height, (uint)width, (uint)size_pitch ,(uint)channel);

			checkCudaErrors(hipDeviceSynchronize());
		}

	}
	
	

	TIMED("hipMemcpyDeviceToHost")
	{
		checkCudaErrors(hipMemcpy2D(dst_image.data, size_width, devPtr_dst, size_pitch,size_width,size_height, hipMemcpyDeviceToHost));

		////memory crashed!below
		//checkCudaErrors(hipMemcpy(dst_image.data, devPtr_dst, size_width*size_height*3, hipMemcpyDeviceToHost));

		checkCudaErrors(hipDeviceSynchronize());
		/*
		for (int i = 0; i < height; i++)
		{
		uchar* data = dst_image.ptr<uchar>(i);
		for (int j = 0; j < width; j++)
		{
		printf("%d,%d,%u,%u,%u\n", i, j, data[3 * j], data[3 * j+1], data[3 * j+2]);
		}
		printf("\n\n");
		}
		*/
	}


	TIMED("hipDeviceSynchronize")
	{
		checkCudaErrors(hipDeviceSynchronize());
	}

	TIMED("cudaCudaFree")
	{
		checkCudaErrors(hipFree(devPtr_src));
		checkCudaErrors(hipFree(devPtr_dst));

	}

	TIMED("hipGetLastError")
	{
		checkCudaErrors(hipGetLastError());
	}
	/*
	for (int i = 0; i < size_height; i++)
	{
		uchar *ptr = dst_image.ptr<uchar>(i);
		for (int j = 0; j < size_width; j++)
		{
			printf("i=%d,j=%d,c=%d,v=%u\n", i, j / 3, j % 3, ptr[j]);
		}
	}
	*/
	
	//cv::namedWindow(titile, cv::WINDOW_NORMAL);// Create a window for display.
	//cv::imshow(titile, dst_image);                   // Show our image inside it.
	cv::imwrite("e:/tmp/" + titile + ".jpg", dst_image);
	//cv::waitKey(0);                                          // Wait for a keystroke in the window
	
	return true;
}