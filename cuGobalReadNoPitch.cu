#include "hip/hip_runtime.h"
#include "functions.h"

__global__ void copyImageNoPitch(uchar * src, uchar * dst, uint imageH, uint imageW, uint channel)
{
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y + threadIdx.y;

	if ((x < imageW) && (y < imageH))
	{
		if (channel == 3)
		{
			uchar* ptr_src = src + channel*(y*imageW + x);
			uchar* ptr_dst = dst + channel*(y*imageW + x);
			ptr_dst[0] = ptr_src[0];
			ptr_dst[1] = ptr_src[1];
			ptr_dst[2] = ptr_src[2];
		}
		else///channel == 4
		{
			uchar4* ptr_src = (uchar4*)(src) + (y*imageW + x);
			uchar4* ptr_dst = (uchar4*)(dst) + (y*imageW + x);
			ptr_dst[0] = ptr_src[0];
		}
	}
}

bool testGlobalReadNoPitch(cv::Mat &image, std::string  titile)
{

	cv::Mat dst_image;
	uchar *devPtr_src, *devPtr_dst;
	size_t size = image.cols*image.rows*sizeof(uchar)* image.channels();

	int width = image.cols;
	int height = image.rows;
	int channel = image.channels();

	if (image.type() == CV_8UC3)
	{
		TIMED("malloc host dst memory")
		{
			dst_image.create(image.rows, image.cols, CV_8UC3);
		}
	}
	else if (image.type() == CV_8UC4)
	{
		TIMED("malloc host dst memory")
		{
			dst_image.create(image.rows, image.cols, CV_8UC4);
		}
	}
	

	



	
	TIMED("malloc device dst memory")
	{
		checkCudaErrors(hipMalloc((void**)&devPtr_src, size));
		checkCudaErrors(hipMalloc((void**)&devPtr_dst, size));
	}

	
	TIMED("hipMemcpyHostToDevice")
	{
		checkCudaErrors(hipMemcpy(devPtr_src, image.data, size, hipMemcpyHostToDevice));
	}

	

	dim3 blockSize(16, 16, 1);
	dim3 gridSize(((uint)width + blockSize.x - 1) / blockSize.x, ((uint)height + blockSize.y - 1) / blockSize.y, 1);

	TIMED("copyImage")
	{
		for (int i = 0; i < 10000; i++)
		{
			copyImageNoPitch <<<gridSize, blockSize >>>(devPtr_src, devPtr_dst,  (uint)height ,(uint)width, (uint)channel);
			checkCudaErrors(hipDeviceSynchronize());
		}
		
	}

	

	TIMED("hipMemcpyDeviceToHost")
	{
		checkCudaErrors(hipMemcpy(dst_image.data, devPtr_dst, size, hipMemcpyDeviceToHost));
		checkCudaErrors(hipDeviceSynchronize());
		/*
		for (int i = 0; i < height; i++)
		{
			uchar* data = dst_image.ptr<uchar>(i);
			for (int j = 0; j < width; j++)
			{
				printf("%d,%d,%u,%u,%u\n", i, j, data[3 * j], data[3 * j+1], data[3 * j+2]);
			}
			printf("\n\n");
		}
		*/
	}


	TIMED("hipDeviceSynchronize")
	{
		checkCudaErrors(hipDeviceSynchronize());
	}

	TIMED("cudaCudaFree")
	{
		checkCudaErrors(hipFree(devPtr_src));
		checkCudaErrors(hipFree(devPtr_dst));

	}

	TIMED("hipGetLastError")
	{
		checkCudaErrors(hipGetLastError());
	}



	//cv::namedWindow(titile, cv::WINDOW_AUTOSIZE);// Create a window for display.
	//cv::imshow(titile, dst_image);                   // Show our image inside it.
	cv::imwrite("e:/tmp/" + titile + ".jpg", dst_image);
	//cv::waitKey(0);                                          // Wait for a keystroke in the window
	return true;
}