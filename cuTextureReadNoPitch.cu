#include "hip/hip_runtime.h"
#include "functions.h"
///type,dim,mode,
texture<uchar, 1, hipReadModeElementType> tex;

texture<uchar4, 1, hipReadModeElementType> tex4;

__global__ void copyImageTexNoPitch(uchar * src, uchar * dst, uint imageH, uint imageW, uint channel)
{
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y + threadIdx.y;

	if ((x < imageW) && (y < imageH))
	{
		if (channel == 3)
		{

			int index = channel * (y*imageW + x);
			//uchar* ptr_src = src + index;
			uchar* ptr_dst = dst + index;
			ptr_dst[0] = tex1Dfetch(tex, index + 0);
			ptr_dst[1] = tex1Dfetch(tex, index + 1);
			ptr_dst[2] = tex1Dfetch(tex, index + 2);
		}
		else ///channel ==4
		{
			int index = (y*imageW + x);
			uchar4* ptr_dst = (uchar4*)dst + index;
			ptr_dst[0] = tex1Dfetch(tex4, index + 0);
		}
	}
}

bool testTexReadNoPitch(cv::Mat &image, std::string  titile)
{

	uchar *devPtr_src, *devPtr_dst;
	size_t size = image.cols*image.rows*sizeof(uchar)* image.channels();
	cv::Mat dst_image;
	///we need a channel desc,we have 1,2,4 channel,no 3!
	hipChannelFormatDesc cf;

	if (image.type() == CV_8UC3)
	{
		TIMED("malloc host dst memory")
		{
			dst_image.create(image.rows, image.cols, CV_8UC3);
		}
		cf = hipCreateChannelDesc<uchar>();
	}
	else if (image.type() == CV_8UC4)
	{
		TIMED("malloc host dst memory")
		{
			dst_image.create(image.rows, image.cols, CV_8UC4);
		}
		cf = hipCreateChannelDesc<uchar4>();
	}
	

	// specify mutable texture reference parameters
	tex.normalized = 0;
	///hipFilterModeLinear is only supported by float!
	tex.filterMode = hipFilterModePoint;
	tex.addressMode[0] = hipAddressModeClamp;

	
	TIMED("malloc device dst memory")
	{
		checkCudaErrors(hipMalloc((void**)&devPtr_src, size));
		checkCudaErrors(hipMalloc((void**)&devPtr_dst, size));
	}
	

	

	
	if (image.type() == CV_8UC3)
	{
		// bind texture reference to array
		checkCudaErrors(hipBindTexture(NULL, &tex, devPtr_src, &cf, size));
	}
	else if (image.type() == CV_8UC4)
	{
		// bind texture reference to array
		checkCudaErrors(hipBindTexture(NULL, &tex4, devPtr_src, &cf, size));
	}



	TIMED("hipMemcpyHostToDevice")
	{
		checkCudaErrors(hipMemcpy(devPtr_src, image.data, size, hipMemcpyHostToDevice));
	}

	int width = image.cols;
	int height = image.rows;
	int channel = image.channels();

	dim3 blockSize(16, 16, 1);
	dim3 gridSize(((uint)width + blockSize.x - 1) / blockSize.x, ((uint)height + blockSize.y - 1) / blockSize.y, 1);

	TIMED("copyImage")
	{
		for (int i = 0; i < 10000; i++)
		{
			copyImageTexNoPitch << <gridSize, blockSize >> >(devPtr_src, devPtr_dst, (uint)height, (uint)width, (uint)channel);
			checkCudaErrors(hipDeviceSynchronize());
		}

	}



	TIMED("hipMemcpyDeviceToHost")
	{
		checkCudaErrors(hipMemcpy(dst_image.data, devPtr_dst, size, hipMemcpyDeviceToHost));
		checkCudaErrors(hipDeviceSynchronize());
		/*
		for (int i = 0; i < height; i++)
		{
		uchar* data = dst_image.ptr<uchar>(i);
		for (int j = 0; j < width; j++)
		{
		printf("%d,%d,%u,%u,%u\n", i, j, data[3 * j], data[3 * j+1], data[3 * j+2]);
		}
		printf("\n\n");
		}
		*/
	}


	TIMED("hipDeviceSynchronize")
	{
		checkCudaErrors(hipDeviceSynchronize());
	}

	TIMED("cudaCudaFree")
	{
		checkCudaErrors(hipFree(devPtr_src));
		checkCudaErrors(hipFree(devPtr_dst));

	}

	TIMED("hipGetLastError")
	{
		checkCudaErrors(hipGetLastError());
	}



	//cv::namedWindow(titile, cv::WINDOW_NORMAL);// Create a window for display.
	//cv::imshow(titile, dst_image);                   // Show our image inside it.
	cv::imwrite("e:/tmp/" + titile + ".jpg", dst_image);
	//cv::waitKey(0);                                          // Wait for a keystroke in the window
	return true;
}