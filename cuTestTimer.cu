#include "functions.h"

StopWatchInterface *timer = NULL;

int initCUDA()
{
	int result = 0;
	result = findCudaDevice(0, NULL);
	return result;
}
void cleanCUDA()
{
	hipDeviceReset();
}

///use to confirm myTimer is good for CUDA timing
bool testTimer1(cv::Mat &image)
{
	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);
	if (image.type() != CV_8UC3)
	{
		return false;
	}
	cv::Mat dst_image(image.rows, image.cols, CV_8UC3);

	sdkStopTimer(&timer);
	//sdkResetTimer(&timer);
	printf("allocate host dst memory time elpse %3.1f ms\n", sdkGetTimerValue(&timer));
	sdkStartTimer(&timer);

	uchar *devPtr;
	size_t size = image.cols*image.rows*sizeof(char)* 3;
	
	checkCudaErrors(hipMalloc((void**) &devPtr, size));
	sdkStopTimer(&timer);
	printf("allocate device dst memory time elpse %3.1f ms\n", sdkGetTimerValue(&timer));
	sdkStartTimer(&timer);


	checkCudaErrors(hipMemcpy(devPtr, image.data, size, hipMemcpyHostToDevice));

	sdkStopTimer(&timer);
	printf("hipMemcpyHostToDevice time elpse %3.1f ms\n", sdkGetTimerValue(&timer));
	sdkStartTimer(&timer);


	checkCudaErrors(hipMemcpy(dst_image.data,devPtr, size, hipMemcpyDeviceToHost));

	sdkStopTimer(&timer);
	printf("hipMemcpyDeviceToHost time elpse %3.1f ms\n", sdkGetTimerValue(&timer));
	sdkStartTimer(&timer);
	
	checkCudaErrors(hipDeviceSynchronize());
	sdkStopTimer(&timer);
	printf("hipDeviceSynchronize time elpse %3.1f ms\n", sdkGetTimerValue(&timer));
	sdkStartTimer(&timer);


	checkCudaErrors(hipFree(devPtr));
	sdkStopTimer(&timer);
	printf("hipFree time elpse %3.1f ms\n", sdkGetTimerValue(&timer));
	sdkStartTimer(&timer);

	checkCudaErrors(hipGetLastError());
	sdkStopTimer(&timer);
	printf("hipGetLastError time elpse %3.1f ms\n", sdkGetTimerValue(&timer));
	
	
	sdkDeleteTimer(&timer);


	cv::namedWindow("dst_image_Display window", cv::WINDOW_AUTOSIZE);// Create a window for display.
	cv::imshow("dst_image_Display window", dst_image);                   // Show our image inside it.

	cv::waitKey(0);                                          // Wait for a keystroke in the window
	return true;
}

bool testTimer2(cv::Mat &image)
{
	
	if (image.type() != CV_8UC3)
	{
		return false;
	}
	cv::Mat dst_image;

	TIMED("malloc host dst memory")
	{
		dst_image.create(image.rows, image.cols, CV_8UC3);
	}

	

	uchar *devPtr;
	size_t size = image.cols*image.rows*sizeof(char)* 3;
	TIMED("malloc device dst memory")
	{
		checkCudaErrors(hipMalloc((void**)&devPtr, size));
	}


	TIMED("hipMemcpyHostToDevice")
	{
		checkCudaErrors(hipMemcpy(devPtr, image.data, size, hipMemcpyHostToDevice));
	}



	TIMED("hipMemcpyDeviceToHost")
	{
		checkCudaErrors(hipMemcpy(dst_image.data, devPtr, size, hipMemcpyDeviceToHost));
		checkCudaErrors(hipDeviceSynchronize());
	}


	TIMED("hipDeviceSynchronize")
	{
		
	}

	TIMED("cudaCudaFree")
	{
		checkCudaErrors(hipFree(devPtr));
	}
	
	TIMED("hipGetLastError")
	{
		checkCudaErrors(hipGetLastError());
	}



	cv::namedWindow("dst_image_Display window", cv::WINDOW_AUTOSIZE);// Create a window for display.
	cv::imshow("dst_image_Display window", dst_image);                   // Show our image inside it.

	cv::waitKey(0);                                          // Wait for a keystroke in the window
	return true;
}

