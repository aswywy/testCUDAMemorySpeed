#include "hip/hip_runtime.h"
#include "functions.h"
#include "functions.h"


texture<uchar, 2, hipReadModeElementType> tex;

texture<uchar4, 2, hipReadModeElementType> tex4;
__global__ void copyImageTexPitch(uchar * src, uchar * dst, uint imageH, uint imageW, uint pitch, uint channel)
{
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y + threadIdx.y;

	if ((x < imageW) && (y < imageH))
	{
		/*
		uchar* ptr_src = src + (y * pitch + 3 * x);
		uchar* ptr_dst = dst + (y * pitch + 3 * x);
		
		ptr_dst[0] = (y * pitch + 3 * x) + 0;
		ptr_dst[1] = (y * pitch + 3 * x) + 1;
		ptr_dst[2] = (y * pitch + 3 * x) + 2;
		
		ptr_dst[0] = ptr_src[0];
		ptr_dst[1] = ptr_src[1];
		ptr_dst[2] = ptr_src[2];
		*/
		if (channel == 3)
		{
			uchar* ptr_dst = dst + (y * pitch + channel * x);
			ptr_dst[0] = tex2D(tex, channel * x + 0, y);
			ptr_dst[1] = tex2D(tex, channel * x + 1, y);
			ptr_dst[2] = tex2D(tex, channel * x + 2, y);

		}
		else //channel ==4
		{
			uchar4 * ptr_dst = (uchar4*)(dst + y*pitch) + x;
			ptr_dst[0] = tex2D(tex4, x, y);
		}

	}
}

bool testTexReadPitch(cv::Mat &image, std::string  titile)
{	
	
	
	uchar *devPtr_src, *devPtr_dst;
	size_t size_width = image.cols*sizeof(uchar)* image.channels();
	size_t size_height = image.rows;
	size_t size_pitch;
	cv::Mat dst_image;

	hipChannelFormatDesc cf;
	



	TIMED("malloc device dst memory,and clear it!!")
	{
		checkCudaErrors(hipMallocPitch((void**)&devPtr_src, &size_pitch, size_width, size_height));
		checkCudaErrors(hipMallocPitch((void**)&devPtr_dst, &size_pitch, size_width, size_height));
		checkCudaErrors(hipMemset2D(devPtr_dst, size_pitch, 0, size_width, size_height));
	}

	if (image.type() == CV_8UC3)
	{
		cf = hipCreateChannelDesc<uchar>();

		// bind texture reference to array
		checkCudaErrors(hipBindTexture2D(NULL, &tex, devPtr_src, &cf,
			size_width, 
			size_height, 
			size_pitch));

		TIMED("malloc host dst memory")
		{
			dst_image.create(image.rows, image.cols, CV_8UC3);
		}
	}
	else if (image.type() == CV_8UC4)
	{
		cf = hipCreateChannelDesc<uchar4>();

		// bind texture reference to array
		checkCudaErrors(hipBindTexture2D(NULL, &tex4, devPtr_src, &cf, 
			size_width/sizeof(uchar4), // in texels
			size_height, // in texels
			size_pitch));
		TIMED("malloc host dst memory")
		{
			dst_image.create(image.rows, image.cols, CV_8UC4);
		}

	}



	tex.normalized = 0;
	tex.filterMode = hipFilterModePoint;
	tex.addressMode[0] = hipAddressModeClamp;
	tex.addressMode[1] = hipAddressModeClamp;

	
	TIMED("hipMemcpyHostToDevice")
	{
		//hipPitchedPtr tep = make_hipPitchedPtr(image.data, size_pitch, size_width, size_height);
		checkCudaErrors(hipMemcpy2D(devPtr_src, size_pitch, image.data, size_width, size_width, size_height, hipMemcpyHostToDevice));
	}
	

	int width = image.cols;
	int height = image.rows;
	int channel = image.channels();

	dim3 blockSize(16, 16, 1);
	dim3 gridSize(((uint)width + blockSize.x - 1) / blockSize.x, ((uint)height + blockSize.y - 1) / blockSize.y, 1);
	//dim3 gridSize(((uint)size_pitch/3 + 1 + blockSize.x - 1) / blockSize.x, ((uint)height + blockSize.y - 1) / blockSize.y, 1);

	TIMED("copyImage")
	{
		for (int i = 0; i < 10000; i++)
		{
			//copyImagePitch << <gridSize, blockSize >> >(devPtr_src, devPtr_dst, (uint)height, (uint)width, (uint)channel);
			copyImageTexPitch << <gridSize, blockSize >> >(devPtr_src, devPtr_dst, (uint)height, (uint)width, (uint)size_pitch, (uint)channel);

			checkCudaErrors(hipDeviceSynchronize());
		}

	}



	TIMED("hipMemcpyDeviceToHost")
	{
		checkCudaErrors(hipMemcpy2D(dst_image.data, size_width, devPtr_dst, size_pitch, size_width, size_height, hipMemcpyDeviceToHost));

		////memory crashed!below
		//checkCudaErrors(hipMemcpy(dst_image.data, devPtr_dst, size_width*size_height*3, hipMemcpyDeviceToHost));

		checkCudaErrors(hipDeviceSynchronize());
		/*
		for (int i = 0; i < height; i++)
		{
		uchar* data = dst_image.ptr<uchar>(i);
		for (int j = 0; j < width; j++)
		{
		printf("%d,%d,%u,%u,%u\n", i, j, data[3 * j], data[3 * j+1], data[3 * j+2]);
		}
		printf("\n\n");
		}
		*/
	}


	TIMED("hipDeviceSynchronize")
	{
		checkCudaErrors(hipDeviceSynchronize());
	}

	TIMED("cudaCudaFree")
	{
		checkCudaErrors(hipFree(devPtr_src));
		checkCudaErrors(hipFree(devPtr_dst));

	}

	TIMED("hipGetLastError")
	{
		checkCudaErrors(hipGetLastError());
	}
	/*
	for (int i = 0; i < size_height; i++)
	{
	uchar *ptr = dst_image.ptr<uchar>(i);
	for (int j = 0; j < size_width; j++)
	{
	printf("i=%d,j=%d,c=%d,v=%u\n", i, j / 3, j % 3, ptr[j]);
	}
	}
	*/

	//cv::namedWindow(titile, cv::WINDOW_NORMAL);// Create a window for display.
	//cv::imshow(titile, dst_image);                   // Show our image inside it.
	cv::imwrite("e:/tmp/" + titile + ".jpg", dst_image);
	//cv::waitKey(0);                                          // Wait for a keystroke in the window

	return true;
}